#include "hip/hip_runtime.h"
#include "CuComponent.cuh"
#include "CuInterpolation.h"

#include <stdio.h>

namespace HBXDef
{

	template<unsigned int T, HBXDef::CudaMalloc_t M >
	__global__	void cutestlag1( HBXDef::CuInterpolation<T, M>* _ArrayIn, HBXDef::UserDefFloat* _outData )
	{
		unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

		while ( idx < MAX_GPUITER )
		{
#ifdef USERDEFSHARED//Ϊ�˿��ƹ����ڴ��С

#endif // USERDEFSHARED//Ϊ�˿��ƹ����ڴ��С

#ifdef WITHTEST	//����������������ָ��Ϊ��������
//		std::cerr<<"����ָ��Ϊ��..."<<std::endl;
#endif  
			HBXDef::UserDefFloat intercord[4];
			for (size_t i = 0; i < RUNCLC; i++)
			{
				intercord[0] = -3.0 + i * 20.0 / RUNCLC + 20.0 * i / MAX_GPUITER;
				intercord[1] = 0.2 + i * 2.0 / RUNCLC + 2.0 * i / MAX_GPUITER;
				intercord[2] = -30 + i * 30.0 / RUNCLC + 30 * i / MAX_GPUITER;
				intercord[3] = -5.0 + 5*i / RUNCLC + i*5 / MAX_GPUITER;

				_ArrayIn[idx].ReadTableData(intercord);
				
			}
			idx += gridDim.x*blockDim.x;
		}
		__syncthreads();
	}




}