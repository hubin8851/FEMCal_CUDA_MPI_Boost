#include "hip/hip_runtime.h"
#ifndef _CUBICTEXTURE_CU_
#define _CUBICTEXTURE_CU_

#include <stdio.h>
#include "HbxDefMacro.h"

#include ""
#include <usercuda/helper_gl.h>
//#include <usercuda/GL/freeglut.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
// Helper functions
#include <helper_functions.h> // CUDA SDK Helper functions
#include <hip/hip_runtime_api.h>// CUDA device initialization helper functions
//#include <helper_cuda_gl.h>   // CUDA device + OpenGL initialization functions
#include "cubicTexture_kernel.cuh"



hipArray *d_Array = 0;




//����Ϊһά�������س�ʼ������
extern "C"
void initTexture1D( size_t _wide, size_t _height, float1* h_data )
{
	hipChannelFormatDesc _channelDesc = hipCreateChannelDesc<float>();
	checkCudaErrors( hipMallocArray(&d_Array, &_channelDesc, _wide, _height) );

	//������������
	tex1d.addressMode[0] = hipAddressModeClamp;//Ѱַģʽ��hipAddressModeClamp��ʾ��������ǯλ��Ѱַ�ļ�ֵ��hipAddressModeWrap���ʾ��������Χ��ֵ���ҽ�֧�ֹ�һ������
	tex1d.addressMode[1] = hipAddressModeClamp;//�Էǹ�һ�������꣬���Ѱַ�����곬���˷�Χ[0��N]������N�����꽫��ǯλ����ΪN-1��
	tex1d.filterMode = hipFilterModeLinear;	//�˲�ģʽ��hipFilterModePoint������ӽ��ĵ㣬hipFilterModeLinear�򷵻����Բ�ֵ����ֵ
	tex1d.normalized = false;
	checkCudaErrors( hipBindTextureToArray(tex1d, d_Array, _channelDesc) );
}

//@_wide:��ά��ֵ��Ŀ��
//@_height����ά��ֵ��ĸ߶�
//@_blckSize�������߳���
//@_gridSize��grid���߳���
//@filter_mode����ֵ����
//@_output���������
extern "C"
void testTexture1D(float1* d_input, size_t _width, size_t _height, dim3 _blckSize, dim3 _gridSize, int filter_mode, HBXDef::UserDefFloat* _output)
{
	initTexture1D(_width, _height, d_input);
	switch (filter_mode)
	{
	case interpolate_t::NEAREST:
		texref.filterMode = hipFilterModePoint;//���þ͸����ĵ�Ĳ�ֵ����
		d_render1D<<<_gridSize, _blckSize>>>( d_input, _width, _height, _output );
		break;
	default:
		break;
	}

}


//����ά���鴦������
//@_wide:���ݿ��
//@_height�����ݸ߶�
//@h_data�������׵�ַ��Ҫ���ַ����
extern "C"
void initTexture2D( size_t _wide, size_t _height, float* h_data )
{
	//���ڴ������ݿ������Դ�
	//	hipChannelFormatDesc _channelDesc = hipCreateChannelDesc(sizeof(HBXDef::UserDefFloat), 0, 0, 0, hipChannelFormatKindFloat);
	hipChannelFormatDesc _channelDesc = hipCreateChannelDesc<float>();


	checkCudaErrors( hipMallocArray(&d_Array, &_channelDesc, _wide, _height) );
	unsigned int tmp_size = _wide *_height *sizeof(HBXDef::UserDefFloat);


	checkCudaErrors( hipMemcpyToArray(d_Array, 0, 0, h_data, tmp_size, hipMemcpyHostToDevice) );
	//free(h_data);

	//������������
	texref.addressMode[0] = hipTextureAddressMode::hipAddressModeBorder;//Ѱַģʽ��hipAddressModeClamp��ʾ��������ǯλ��Ѱַ�ļ�ֵ��hipAddressModeWrap���ʾ��������Χ��ֵ���ҽ�֧�ֹ�һ������
	texref.addressMode[1] = hipTextureAddressMode::hipAddressModeBorder;//�Էǹ�һ�������꣬���Ѱַ�����곬���˷�Χ[0��N]������N�����꽫��ǯλ����ΪN-1��
	texref.filterMode = hipFilterModeLinear;	//�˲�ģʽ��hipFilterModePoint������ӽ��ĵ㣬hipFilterModeLinear�򷵻����Բ�ֵ����ֵ
	texref.normalized = false;

	checkCudaErrors( hipBindTextureToArray(texref, d_Array, _channelDesc) );

}


//@_wide:��ά��ֵ��Ŀ��
//@_height����ά��ֵ��ĸ߶�
//@_blckSize�������߳���
//@_gridSize��grid���߳���
//@filter_mode����ֵ����
//@_output���������
extern "C"
void testTexture2D(float2* d_input, size_t _width, size_t _height, dim3 _blckSize, dim3 _gridSize, int filter_mode, HBXDef::UserDefFloat* _output)
{
	float* interpotdata = new float[_width*_height];
	for (int i = 0; i < _height; i++)
	{
		for (int j = 0; j < _width; j++)
		{
			interpotdata[i*_width+j] = i*_width+j;
		}
	}

	initTexture2D(_width, _height, interpotdata);
	switch (filter_mode)
	{
	case interpolate_t::NEAREST:
//		texref.filterMode = hipFilterModePoint;//���þ͸����ĵ�Ĳ�ֵ����
		d_render2D<<<_gridSize, _blckSize>>>( d_input, _width, _height, _output );
		break;
	default:
		break;
	}
	hipDeviceSynchronize();
	std::cerr<<hipGetErrorName(hipGetLastError())<<std::endl;
}


#endif