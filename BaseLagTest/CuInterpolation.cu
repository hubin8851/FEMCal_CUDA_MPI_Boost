#include "hip/hip_runtime.h"
#include <stdio.h>

#include "XMLManage.h"
#include "HbxDefMacro.h"
#include "CuComponent.cuh"

#include "CuInterpolation.cuh"


namespace HBXDef
{

// 	template<unsigned int T, HBXDef::CudaMalloc_t M >
// 	__global__	void cutestlag1( HBXDef::CuInterpolation<T, M>* _ArrayIn, HBXDef::UserDefFloat* _outData )
// 	{
// 		using namespace HBXDef;
// 		unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
// 
// 		while ( idx < MAX_GPUITER )
// 		{
// #ifdef USERDEFSHARED//Ϊ�˿��ƹ����ڴ��С
// 
// #endif // USERDEFSHARED//Ϊ�˿��ƹ����ڴ��С
// 
// #ifdef WITHTEST	//����������������ָ��Ϊ��������
// //		std::cerr<<"����ָ��Ϊ��..."<<std::endl;
// 
// 			UserDefFloat intercord[4];
// 			for (size_t i = 0; i < RUNCLC; i++)
// 			{
// 				intercord[0] = -3.0 + i * 20.0 / RUNCLC + 20.0 * i / MAX_GPUITER;
// 				intercord[1] = 0.2 + i * 2.0 / RUNCLC + 2.0 * i / MAX_GPUITER;
// 				intercord[2] = -30 + i * 30.0 / RUNCLC + 30 * i / MAX_GPUITER;
// 				intercord[3] = -5.0 + 5*i / RUNCLC + i*5 / MAX_GPUITER;
// 
// 				_outData[i + RUNCLC*idx] = _ArrayIn[idx].ReadTableData(intercord);
// 				
// 			}
// 			idx += gridDim.x*blockDim.x;
// #endif  //WITHTEST
// 		}
// 		__syncthreads();
// 	}



}



extern "C"
void testlag()
{
	using namespace HBXDef;
	CXMLManage	g_Xml;		//���ڶ�ȡ��Ԫ���Ե�xml
	g_Xml.ReadAeroCoef("F:\\data from HBX_phd\\database\\x33��������\\CA_d0.xml");

	const unsigned int g_T = 2;
	typedef	HBXDef::CBaseLag<g_T, HBXDef::CudaMalloc_t::UNIFIEDMEM>	_samelag;
	typedef	HBXDef::CuInterpolation<g_T, HBXDef::CudaMalloc_t::UNIFIEDMEM>	_sameCulag;
	float elapsedTime;

	HBXDef::UserDefFloat* g_opData = nullptr;
#ifdef WITHTEST
	std::cout<<hipGetErrorName( hipMallocManaged(&g_opData, RUNCLC * MAX_GPUITER* sizeof(HBXDef::UserDefFloat)) )<<std::endl;
#endif // WITHTEST


	_samelag*	g_BaseLag = new _samelag( &g_Xml.GetAeroTable("CA_d0"), 0 );	//���Բ�ֵ
	g_BaseLag->SetBlkId(0);
	g_BaseLag->Initial();

	_sameCulag*	g_culag = nullptr;
	std::cout<<sizeof(_sameCulag)<<std::endl;
	std::cout<<hipGetErrorName( hipMalloc(&g_culag, MAX_GPUITER* sizeof(_sameCulag)) )<<std::endl;
	//	g_culag = (_sameCulag*)malloc(MAX_GPUITER* sizeof(_sameCulag));
	if (!g_culag)
	{
		cout << "�ڴ�������" << endl;
		return;
	}

	for (int i = 0; i < MAX_GPUITER; i++)
	{
//		g_culag[i].Initial(*g_BaseLag);
	}

	//�����㷨
	clock_t _beg, _end;
	HBXDef::UserDefFloat _rlt;
	_beg = clock();
	HBXDef::UserDefFloat  g_dataIn[4];
	for (size_t i = 0; i < MAX_GPUITER; i++) {
		for (size_t j = 0; j < RUNCLC; j++) {
			g_dataIn[0] = -3.0 + j * 20.0 / RUNCLC + 20.0 * i / MAX_GPUITER;
			g_dataIn[1] = 0.2 + j * 2.0 / RUNCLC + 2.0 * i / MAX_GPUITER;
			g_dataIn[2] = -30 + j * 30.0 / RUNCLC + 30 * i / MAX_GPUITER;
			g_dataIn[3] = -5.0 + 5*j / RUNCLC + i*5 / MAX_GPUITER;
			_rlt = g_BaseLag->ReadTableData(g_dataIn);
		}
	}
	_end = clock();
	cout << "CPU time:" << _end - _beg << endl;

#ifdef WITHTEST
	std::cout<<_rlt<<std::endl;
#else
	std::cout<<_rlt<<std::endl;
#endif // WITHTEST
	//�����㷨

	cuerror(hipSetDevice(0));

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);//��ʼ��ʱ

#ifdef WITHTEST //��������
	cutestlag1<g_T, CudaMalloc_t::UNIFIEDMEM><<<GRIDSIZE, BLOCKSIZE>>>(g_culag, g_opData);
	hipEventRecord(stop, 0);//��ʱ����
	std::cerr<<hipGetErrorName(hipGetLastError())<<endl;
	hipDeviceSynchronize();
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);//��ȡ��ʱ
#else
	cutestlag1<g_T, CudaMalloc_t::UNIFIEDMEM><<<GRIDSIZE, BLOCKSIZE, MAX_SHARED, 0 >>>(g_culag, nullptr);
	hipEventRecord(stop, 0);//��ʱ����
	std::cerr<<hipGetErrorString(hipGetLastError())<<endl;
	hipDeviceSynchronize();
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);//��ȡ��ʱ
#endif // WITHTEST


	std::cout << "time eclaps:" << elapsedTime << std::endl;
	system("pause");
}